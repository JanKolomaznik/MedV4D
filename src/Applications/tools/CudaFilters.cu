#include "hip/hip_runtime.h"
#include "CudaUtils.cuh"
#include "Imaging/ImageRegion.h"


template< typename TElement >
struct SobelFilter3DFtor
{
	typedef typename TypeTraits< TElement >::SignedClosestType SignedElement;

	SobelFilter3DFtor( TElement aThreshold = 0 ): threshold( aThreshold ), radius( make_int3( 1, 1, 1 ) )
	{}

	__device__ TElement
	operator()( TElement data[], uint idx, uint syStride, uint szStride )
	{
		TElement val1 = abs( (2*static_cast<SignedElement>(data[idx+1]) - 2*static_cast<SignedElement>(data[idx-1])) 
			+ (static_cast<SignedElement>(data[idx+1 + syStride]) - static_cast<SignedElement>(data[idx-1 + syStride]))
			+ (static_cast<SignedElement>(data[idx+1 - syStride]) - static_cast<SignedElement>(data[idx-1 - syStride])) );
		TElement val2 = abs( (2*static_cast<SignedElement>(data[idx+syStride]) - 2*static_cast<SignedElement>(data[idx-syStride])) 
			+ (static_cast<SignedElement>(data[idx+1 + syStride]) - static_cast<SignedElement>(data[idx+1 - syStride]))
			+ (static_cast<SignedElement>(data[idx-1 + syStride]) - static_cast<SignedElement>(data[idx-1 - syStride])) );
		TElement val3 = abs( (2*static_cast<SignedElement>(data[idx+szStride]) - 2*static_cast<SignedElement>(data[idx-szStride])) 
			+ (static_cast<SignedElement>(data[idx + syStride + szStride]) - static_cast<SignedElement>(data[idx + syStride - szStride]))
			+ (static_cast<SignedElement>(data[idx - syStride + szStride]) - static_cast<SignedElement>(data[idx - syStride - szStride])) );
		TElement val4 = abs( (2*static_cast<SignedElement>(data[idx + syStride]) - 2*static_cast<SignedElement>(data[idx-syStride])) 
			+ (static_cast<SignedElement>(data[idx + syStride + szStride]) - static_cast<SignedElement>(data[idx - syStride + szStride]))
			+ (static_cast<SignedElement>(data[idx + syStride - szStride]) - static_cast<SignedElement>(data[idx - syStride - szStride])) );
		TElement val5 = abs( (2*static_cast<SignedElement>(data[idx+1]) - 2*static_cast<SignedElement>(data[idx-1])) 
			+ (static_cast<SignedElement>(data[idx + 1 + szStride]) - static_cast<SignedElement>(data[idx-1 + szStride]))
			+ (static_cast<SignedElement>(data[idx + 1 - szStride]) - static_cast<SignedElement>(data[idx-1 - szStride])) );
		TElement val6 = abs( (2*static_cast<SignedElement>(data[idx+szStride]) - 2*static_cast<SignedElement>(data[idx-szStride])) 
			+ (static_cast<SignedElement>(data[idx + 1 + szStride]) - static_cast<SignedElement>(data[idx +1 - szStride]))
			+ (static_cast<SignedElement>(data[idx - 1 + szStride]) - static_cast<SignedElement>(data[idx -1 - szStride])) );
		TElement result = val1 + val2 + val3 + val4 + val5 +val6;
		return result > threshold ? result : 0;
	}

	TElement threshold;
	int3 radius;
};

template< typename TElement >
struct LocalMinima3DFtor
{
	LocalMinima3DFtor(): radius( make_int3( 1, 1, 1 ) )
	{}

	__device__ uint8
	operator()( TElement data[], uint idx, uint syStride, uint szStride )
	{
		TElement value = data[idx];
		for ( int i = idx-1; i <= idx+1; ++i ) {
			for ( int j = i-syStride; j <= i+syStride; j+=syStride ) {
				for ( int k = j-szStride; k <= j+szStride; k+=szStride ) {
					value = min( data[k], value );
				}
			}
		}
		return value >= data[idx] ? 255 : 0;
	}
	int3 radius;
};


__device__ int lutUpdated;

#define min_valid(a, b) (a < b ? a == 0 ? b : a : b == 0 ? a : b)
__device__ uint32
ValidMin( uint32 data[], uint idx, uint syStride, uint szStride )
{
	uint32 value1 = min_valid( data[idx-1], data[idx+1] );
	uint32 value2 = min_valid( data[idx-syStride], data[idx+syStride] );
	uint32 value3 = min_valid( data[idx-szStride], data[idx+szStride] );
	uint32 value = min_valid( value1, value2 );
	return min_valid( value, value3 );
}

__global__ void 
CopyMask( Buffer3D< uint8 > inBuffer, Buffer3D< uint32 > outBuffer )
{ 
	uint blockId = __mul24(blockIdx.y, gridDim.x) + blockIdx.x;
	int idx = blockId * blockDim.x + threadIdx.x;

	if ( idx < inBuffer.mLength ) {
		outBuffer.mData[idx] = inBuffer.mData[idx]!=0 ? idx+1 : 0;
	}
}

__global__ void 
InitLut( Buffer3D< uint32 > outBuffer, Buffer1D< uint32 > lut )
{ 
	uint blockId = __mul24(blockIdx.y, gridDim.x) + blockIdx.x;
	int idx = blockId * blockDim.x + threadIdx.x;

	if ( idx < outBuffer.mLength ) {
		lut.mData[idx] = outBuffer.mData[idx];// = outBuffer.mData[idx] != 0 ? idx+1 : 0;
	}
}

__global__ void 
UpdateLut( Buffer3D< uint32 > buffer, Buffer1D< uint32 > lut )
{ 
	uint blockId = __mul24(blockIdx.y, gridDim.x) + blockIdx.x;
	int idx = blockId * blockDim.x + threadIdx.x;
	uint32 label, ref;
	if ( idx < buffer.mLength ) {
		label = buffer.mData[idx];

		if (label == idx+1) {		
			ref = label-1;
			label = lut.mData[idx];
			while (ref != label-1) {
				ref = label-1;
				label = lut.mData[ref];
			}
			lut.mData[idx] = label;
		}
	}
}

__global__ void 
UpdateLabels( Buffer3D< uint32 > buffer, Buffer1D< uint32 > lut )
{
	uint blockId = __mul24(blockIdx.y, gridDim.x) + blockIdx.x;
	int idx = blockId * blockDim.x + threadIdx.x;

	if ( idx < buffer.mLength ) {
		uint label = buffer.mData[idx];
		if ( label > 0 ) {
			buffer.mData[idx] = lut.mData[label-1];
		}
	}
}

__global__ void 
ScanImage( Buffer3D< uint32 > buffer, Buffer1D< uint32 > lut, int3 blockResolution )
{
	__shared__ uint32 data[MAX_SHARED_MEMORY];
	
	int3 radius = make_int3(1,1,1);
	uint syStride = blockDim.x+2*radius.x;
	uint szStride = (blockDim.x+2*radius.x) * (blockDim.y+2*radius.y);

	uint3 size = buffer.mSize;
	int3 strides = buffer.mStrides;
	uint blockId = __mul24(blockIdx.y, gridDim.x) + blockIdx.x;
	int3 blockCoordinates = GetBlockCoordinates ( blockResolution, blockId );
	int3 blockOrigin = GetBlockOrigin( blockDim, blockCoordinates );
	int3 coordinates = blockOrigin;
	//uint tid = threadIdx.x + blockDim.x * threadIdx.y + blockDim.x * blockDim.y * threadIdx.z;
	coordinates.x += threadIdx.x;
	coordinates.y += threadIdx.y;
	coordinates.z += threadIdx.z;
	bool projected = ProjectionToInterval( coordinates, make_int3(0,0,0), make_int3( size.x, size.y, size.z ) );
	
	int idx = coordinates.x * strides.x + coordinates.y * strides.y + coordinates.z * strides.z;
	uint sidx = (threadIdx.y+radius.y) * syStride + (threadIdx.z+radius.z) * szStride + threadIdx.x + radius.x;
	data[sidx] = buffer.mData[ idx ];
	
	uint3 sIdx;
	int3 mCoordinates = blockOrigin;
	switch( threadIdx.z ) {
	case 0:
		sIdx.x = threadIdx.x + radius.x;
		sIdx.y = threadIdx.y + radius.y;
		sIdx.z = 0;
		break;
	case 1:
		sIdx.x = threadIdx.x + radius.x;
		sIdx.y = threadIdx.y + radius.y;
		sIdx.z = blockDim.z + radius.z;
		break;
	case 2:
		sIdx.x = threadIdx.x + radius.x;
		sIdx.y = 0;
		sIdx.z = threadIdx.y + radius.z;
		break;
	case 3:
		sIdx.x = threadIdx.x + radius.x;
		sIdx.y = blockDim.y + radius.y;
		sIdx.z = threadIdx.y + radius.z;
		break;
	case 4:
		sIdx.x = 0;
		sIdx.y = threadIdx.y + radius.y;
		sIdx.z = threadIdx.x + radius.z;
		break;
	case 5:
		sIdx.x = blockDim.x + radius.x;
		sIdx.y = threadIdx.y + radius.y;
		sIdx.z = threadIdx.x + radius.z;
		break;
	case 6:
		if ( threadIdx.y < 4 ) {
			sIdx.x = threadIdx.x + radius.x;
			sIdx.y = (threadIdx.y & 1)*(blockDim.y + radius.y);
			sIdx.z = (threadIdx.y >> 1)*(blockDim.z + radius.z);
		} else {
			sIdx.x = ((threadIdx.y-4) >> 1)*(blockDim.x + radius.x);
			sIdx.y = threadIdx.x + radius.x;
			sIdx.z = (threadIdx.y & 1)*(blockDim.z + radius.z);
		}
		break;
	case 7:
		if ( threadIdx.y < 4 ) {
			sIdx.x = (threadIdx.y & 1)*(blockDim.x + radius.x);
			sIdx.y = ((threadIdx.y) >> 1)*(blockDim.y + radius.y);
			sIdx.z = threadIdx.x + radius.z;
		} else {	
			sIdx.x = threadIdx.x < 4 ? 0 : (blockDim.x + radius.x);
			sIdx.y = (threadIdx.x >> 1) & 1 ? 0 : (blockDim.y + radius.y);
			sIdx.z = threadIdx.x & 1 ? 0 : (blockDim.z + radius.z);
		}
		break;
	default:
		break;
	}
	mCoordinates.x += sIdx.x - radius.x;
	mCoordinates.y += sIdx.y - radius.y;
	mCoordinates.z += sIdx.z - radius.z;
	ProjectionToInterval( mCoordinates, make_int3(0,0,0), make_int3( size.x, size.y, size.z ) );
	data[sIdx.y*syStride + sIdx.z*szStride + sIdx.x] = buffer.mData[ mCoordinates.x * strides.x + mCoordinates.y * strides.y + mCoordinates.z * strides.z ];

	__syncthreads();

	if( !projected ) {
		uint32 current = data[sidx];
		if ( current != 0 ) {
			uint32 minLabel = ValidMin( data, sidx, syStride, szStride );
			if ( minLabel < current && minLabel != 0) {
				lut.mData[current-1] = minLabel < lut.mData[current-1] ? minLabel : lut.mData[current-1];
				lutUpdated = 1;
			}
		}
	}
}

void
ConnectedComponentLabeling3D( M4D::Imaging::MaskRegion3D input, M4D::Imaging::ImageRegion< uint32, 3 > output )
{
	int lutUpdated = 0;
	Buffer3D< uint8 > inBuffer = CudaBuffer3DFromImageRegionCopy( input );
	Buffer3D< uint32 > outBuffer = CudaBuffer3DFromImageRegion( output );
	int3 radius = make_int3( 1, 1, 1 );


	dim3 blockSize1D( 512 );
	dim3 gridSize1D( (inBuffer.mLength + 64*blockSize1D.x - 1) / (64*blockSize1D.x) , 64 );
	
	dim3 blockSize3D( 8, 8, 8 );
	int3 blockResolution3D = GetBlockResolution( inBuffer.mSize, blockSize3D, make_int3(0,0,0) );
	dim3 gridSize3D( blockResolution3D.x * blockResolution3D.y, blockResolution3D.z, 1 );

	M4D::Common::Clock clock;

	CheckCudaErrorState( "Before execution" );
	CopyMask<<< gridSize1D, blockSize1D >>>( inBuffer, outBuffer );
	CheckCudaErrorState( "After CopyMask()" );
	hipFree( inBuffer.mData );

	Buffer1D< uint32 > lut = CudaAllocateBuffer<uint32>( outBuffer.mLength ); 

	InitLut<<< gridSize1D, blockSize1D >>>( outBuffer, lut );
	CheckCudaErrorState( "After InitLut()" );
	ScanImage<<< gridSize3D, blockSize3D >>>( 
					outBuffer, 
					lut,
					blockResolution3D
					);

	CheckCudaErrorState( "Before iterations" );
	hipMemcpyFromSymbol( &lutUpdated, HIP_SYMBOL("lutUpdated"), sizeof(int), 0, hipMemcpyDeviceToHost );
	while (lutUpdated != 0) {
		LOG( ">" );
                hipMemcpyToSymbol(HIP_SYMBOL( "lutUpdated"), &(lutUpdated = 0), sizeof(int), 0, hipMemcpyHostToDevice );

		UpdateLut<<< gridSize1D, blockSize1D >>>( outBuffer, lut );
		UpdateLabels<<< gridSize1D, blockSize1D >>>( outBuffer, lut );

		ScanImage<<< gridSize3D, blockSize3D >>>( 
					outBuffer, 
					lut,
					blockResolution3D
					);
		hipMemcpyFromSymbol( &lutUpdated, HIP_SYMBOL("lutUpdated"), sizeof(int), 0, hipMemcpyDeviceToHost );
		CheckCudaErrorState( "End of iteration" );
	}
	hipDeviceSynchronize();
	D_PRINT( "Computations took " << clock.SecondsPassed() )

	hipMemcpy(output.GetPointer(), outBuffer.mData, outBuffer.mLength * sizeof(uint32), hipMemcpyDeviceToHost );
	CheckCudaErrorState( "Copy back" );
	hipFree( outBuffer.mData );
	hipFree( lut.mData );
}


__device__ int wshedUpdated;

template< typename TEType >
__global__ void 
InitWatershedBuffers( Buffer3D< uint32 > labeledRegionsBuffer, Buffer3D< TEType > tmpBuffer, TEType infinity )
{ 
	uint blockId = __mul24(blockIdx.y, gridDim.x) + blockIdx.x;
	int idx = blockId * blockDim.x + threadIdx.x;

	if ( idx < tmpBuffer.mLength ) {
		tmpBuffer.mData[idx] = labeledRegionsBuffer.mData[idx] == 0 ? infinity : 0;
	}
}

template< typename TInEType, typename TTmpEType >
__global__ void 
WShedEvolution( Buffer3D< uint32 > labeledRegionsBuffer, Buffer3D< TInEType > inputBuffer, Buffer3D< TTmpEType > tmpBuffer, int3 blockResolution, TTmpEType infinity )
{
	__shared__ uint32 labels[MAX_SHARED_MEMORY];
	__shared__ TTmpEType tmpValues[MAX_SHARED_MEMORY];
	
	int3 radius = make_int3(1,1,1);
	uint syStride = blockDim.x+2*radius.x;
	uint szStride = (blockDim.x+2*radius.x) * (blockDim.y+2*radius.y);

	uint3 size = inputBuffer.mSize;
	int3 strides = inputBuffer.mStrides;
	uint blockId = __mul24(blockIdx.y, gridDim.x) + blockIdx.x;
	int3 blockCoordinates = GetBlockCoordinates ( blockResolution, blockId );
	int3 blockOrigin = GetBlockOrigin( blockDim, blockCoordinates );
	int3 coordinates = blockOrigin;
	//uint tid = threadIdx.x + blockDim.x * threadIdx.y + blockDim.x * blockDim.y * threadIdx.z;
	coordinates.x += threadIdx.x;
	coordinates.y += threadIdx.y;
	coordinates.z += threadIdx.z;
	bool projected = ProjectionToInterval( coordinates, make_int3(0,0,0), make_int3( size.x, size.y, size.z ) );
	
	int idx = coordinates.x * strides.x + coordinates.y * strides.y + coordinates.z * strides.z;
	uint sidx = (threadIdx.y+radius.y) * syStride + (threadIdx.z+radius.z) * szStride + threadIdx.x + radius.x;
	labels[sidx] = labeledRegionsBuffer.mData[ idx ];
	tmpValues[sidx] = tmpBuffer.mData[ idx ];
	
	uint3 sIdx;
	int3 mCoordinates = blockOrigin;
	switch( threadIdx.z ) {
	case 0:
		sIdx.x = threadIdx.x + radius.x;
		sIdx.y = threadIdx.y + radius.y;
		sIdx.z = 0;
		break;
	case 1:
		sIdx.x = threadIdx.x + radius.x;
		sIdx.y = threadIdx.y + radius.y;
		sIdx.z = blockDim.z + radius.z;
		break;
	case 2:
		sIdx.x = threadIdx.x + radius.x;
		sIdx.y = 0;
		sIdx.z = threadIdx.y + radius.z;
		break;
	case 3:
		sIdx.x = threadIdx.x + radius.x;
		sIdx.y = blockDim.y + radius.y;
		sIdx.z = threadIdx.y + radius.z;
		break;
	case 4:
		sIdx.x = 0;
		sIdx.y = threadIdx.y + radius.y;
		sIdx.z = threadIdx.x + radius.z;
		break;
	case 5:
		sIdx.x = blockDim.x + radius.x;
		sIdx.y = threadIdx.y + radius.y;
		sIdx.z = threadIdx.x + radius.z;
		break;
	case 6:
		if ( threadIdx.y < 4 ) {
			sIdx.x = threadIdx.x + radius.x;
			sIdx.y = (threadIdx.y & 1)*(blockDim.y + radius.y);
			sIdx.z = (threadIdx.y >> 1)*(blockDim.z + radius.z);
		} else {
			sIdx.x = ((threadIdx.y-4) >> 1)*(blockDim.x + radius.x);
			sIdx.y = threadIdx.x + radius.x;
			sIdx.z = (threadIdx.y & 1)*(blockDim.z + radius.z);
		}
		break;
	case 7:
		if ( threadIdx.y < 4 ) {
			sIdx.x = (threadIdx.y & 1)*(blockDim.x + radius.x);
			sIdx.y = ((threadIdx.y) >> 1)*(blockDim.y + radius.y);
			sIdx.z = threadIdx.x + radius.z;
		} else {	
			sIdx.x = threadIdx.x < 4 ? 0 : (blockDim.x + radius.x);
			sIdx.y = (threadIdx.x >> 1) & 1 ? 0 : (blockDim.y + radius.y);
			sIdx.z = threadIdx.x & 1 ? 0 : (blockDim.z + radius.z);
		}
		break;
	default:
		break;
	}
	mCoordinates.x += sIdx.x - radius.x;
	mCoordinates.y += sIdx.y - radius.y;
	mCoordinates.z += sIdx.z - radius.z;
	ProjectionToInterval( mCoordinates, make_int3(0,0,0), make_int3( size.x, size.y, size.z ) );

	labels[sIdx.y*syStride + sIdx.z*szStride + sIdx.x] = labeledRegionsBuffer.mData[ mCoordinates.x * strides.x + mCoordinates.y * strides.y + mCoordinates.z * strides.z ];
	tmpValues[sIdx.y*syStride + sIdx.z*szStride + sIdx.x] = tmpBuffer.mData[ mCoordinates.x * strides.x + mCoordinates.y * strides.y + mCoordinates.z * strides.z ];

	__syncthreads();

	if( !projected ) {
		int minIdx = -1;
		int counter = 0;
		int minCounter = -1;
		TInEType value = inputBuffer.mData[ idx ];
		TTmpEType minVal = max( tmpValues[ sidx ] - value,TTmpEType(0) );
		TTmpEType originalMinVal = minVal;
		for ( int i = sidx-1; i <= sidx+1; ++i ) {
			for ( int j = i-syStride; j <= i+syStride; j+=syStride ) {
				for ( int k = j-szStride; k <= j+szStride; k+=szStride ) {
					if( tmpValues[ k ] < minVal ) {
						minVal = tmpValues[ k ];
						minIdx = k;

						minCounter = counter;
					}
					++counter;
				}
			}
		}
		if( minIdx != -1 ) {
			labeledRegionsBuffer.mData[ idx ] = labels[ minIdx ];
			tmpBuffer.mData[ idx ] = tmpValues[minIdx] + value;
			wshedUpdated = 1;
		}
	}
}

#include "Imaging/Image.h"
#include "Imaging/ImageFactory.h"


template< typename TEType >
void
WatershedTransformation3D( M4D::Imaging::ImageRegion< uint32, 3 > aLabeledMarkerRegions, M4D::Imaging::ImageRegion< TEType, 3 > aInput, M4D::Imaging::ImageRegion< uint32, 3 > aOutput )
{
	typedef typename TypeTraits< TEType >::SignedClosestType SignedElement;
	int wshedUpdated = 1;
	Buffer3D< uint32 > labeledRegionsBuffer = CudaBuffer3DFromImageRegionCopy( aLabeledMarkerRegions );
	Buffer3D< TEType > inputBuffer = CudaBuffer3DFromImageRegionCopy( aInput );
	Buffer3D< SignedElement > tmpBuffer = CudaPrepareBuffer<SignedElement>( aInput.GetSize() );
	int3 radius = make_int3( 1, 1, 1 );


	dim3 blockSize1D( 512 );
	dim3 gridSize1D( (inputBuffer.mLength + 64*blockSize1D.x - 1) / (64*blockSize1D.x) , 64 );

	dim3 blockSize3D( 8, 8, 8 );
	int3 blockResolution3D = GetBlockResolution( inputBuffer.mSize, blockSize3D, make_int3(0,0,0) );
	dim3 gridSize3D( blockResolution3D.x * blockResolution3D.y, blockResolution3D.z, 1 );

	M4D::Common::Clock clock;
	D_PRINT( "InitWatershedBuffers()" );
	InitWatershedBuffers<<< gridSize1D, blockSize1D >>>( labeledRegionsBuffer, tmpBuffer, TypeTraits<SignedElement>::Max );

	unsigned i = 0;
	while (wshedUpdated != 0 && i < 51) {
		hipMemcpyToSymbol(HIP_SYMBOL( "wshedUpdated"), &(wshedUpdated = 0), sizeof(int), 0, hipMemcpyHostToDevice );

		//D_PRINT( "WShedEvolution()" );
		WShedEvolution<<< gridSize3D, blockSize3D >>>( 
					labeledRegionsBuffer,
				       	inputBuffer,	
					tmpBuffer,
					blockResolution3D, 
					TypeTraits<SignedElement>::Max
					);

		hipMemcpyFromSymbol( &wshedUpdated, HIP_SYMBOL("wshedUpdated"), sizeof(int), 0, hipMemcpyDeviceToHost );
		++i;
	}

	hipDeviceSynchronize();
	D_PRINT( "Computations took " << clock.SecondsPassed() << " and " << i << " iterations" )

	hipMemcpy(aOutput.GetPointer(), labeledRegionsBuffer.mData, labeledRegionsBuffer.mLength * sizeof(uint32), hipMemcpyDeviceToHost );
	hipFree( labeledRegionsBuffer.mData );
	hipFree( inputBuffer.mData );


	typename M4D::Imaging::Image< SignedElement, 3 >::Ptr tmpDebugImage = M4D::Imaging::ImageFactory::CreateEmptyImageFromExtents< SignedElement, 3 >( aLabeledMarkerRegions.GetMinimum(), aLabeledMarkerRegions.GetMaximum(), aLabeledMarkerRegions.GetElementExtents() );
	hipMemcpy(tmpDebugImage->GetRegion().GetPointer(), tmpBuffer.mData, labeledRegionsBuffer.mLength * sizeof(SignedElement), hipMemcpyDeviceToHost );
	M4D::Imaging::ImageFactory::DumpImage( "Intermediate.dump", *tmpDebugImage );

	hipFree( tmpBuffer.mData );
}

template< typename RegionType >
void
Sobel3D( RegionType input, RegionType output, typename RegionType::ElementType threshold )
{
	typedef typename RegionType::ElementType TElement;
	typedef Buffer3D< TElement > Buffer;

	Buffer inBuffer = CudaBuffer3DFromImageRegionCopy( input );
	Buffer outBuffer = CudaBuffer3DFromImageRegion( output );

	SobelFilter3DFtor< TElement > filter( threshold );
	//int3 radius = filter.radius;

	dim3 blockSize( 8, 8, 8 );
	int3 blockResolution = GetBlockResolution( inBuffer.mSize, blockSize, make_int3(0,0,0) );
	dim3 gridSize( blockResolution.x * blockResolution.y, blockResolution.z, 1 );

	M4D::Common::Clock clock;
	CheckCudaErrorState( "Before kernel execution" );
	FilterKernel3D< TElement, TElement, SobelFilter3DFtor< TElement > >
		<<< gridSize, blockSize >>>( 
					inBuffer, 
					outBuffer, 
					blockResolution,
					filter
					);
	hipDeviceSynchronize();
	CheckCudaErrorState( "After kernel execution" );
	D_PRINT( "Computations took " << clock.SecondsPassed() )

	hipMemcpy(output.GetPointer(), outBuffer.mData, outBuffer.mLength * sizeof(TElement), hipMemcpyDeviceToHost );
	CheckCudaErrorState( "Copy back" );
	hipFree( inBuffer.mData );
	hipFree( outBuffer.mData );
	CheckCudaErrorState( "Free memory" );
}


template< typename RegionType >
void
LocalMinima3D( RegionType input, M4D::Imaging::MaskRegion3D output )
{
	typedef typename RegionType::ElementType TElement;
	
	Buffer3D< TElement > inBuffer = CudaBuffer3DFromImageRegionCopy( input );
	Buffer3D< uint8 > outBuffer = CudaBuffer3DFromImageRegion( output );

	LocalMinima3DFtor< TElement > filter;
	//int3 radius = filter.radius;

	dim3 blockSize( 8, 8, 8 );
	int3 blockResolution = GetBlockResolution( inBuffer.mSize, blockSize, make_int3(0,0,0) );
	dim3 gridSize( blockResolution.x * blockResolution.y, blockResolution.z, 1 );

	M4D::Common::Clock clock;
	CheckCudaErrorState( "Before kernel execution" );
	FilterKernel3D< TElement, uint8, LocalMinima3DFtor< TElement > >
		<<< gridSize, blockSize >>>( 
					inBuffer, 
					outBuffer, 
					blockResolution,
					filter
					);
	hipDeviceSynchronize();
	CheckCudaErrorState( "After kernel execution" );
	D_PRINT( "Computations took " << clock.SecondsPassed() )

	hipMemcpy(output.GetPointer(), outBuffer.mData, outBuffer.mLength * sizeof(uint8), hipMemcpyDeviceToHost );
	CheckCudaErrorState( "Copy back" );
	hipFree( inBuffer.mData );
	hipFree( outBuffer.mData );
	CheckCudaErrorState( "Free memory" );
}




template void Sobel3D( M4D::Imaging::ImageRegion< int8, 3 > input, M4D::Imaging::ImageRegion< int8, 3 > output, int8 threshold );
template void Sobel3D( M4D::Imaging::ImageRegion< uint8, 3 > input, M4D::Imaging::ImageRegion< uint8, 3 > output, uint8 threshold );
template void Sobel3D( M4D::Imaging::ImageRegion< int16, 3 > input, M4D::Imaging::ImageRegion< int16, 3 > output, int16 threshold );
template void Sobel3D( M4D::Imaging::ImageRegion< uint16, 3 > input, M4D::Imaging::ImageRegion< uint16, 3 > output, uint16 threshold );
template void Sobel3D( M4D::Imaging::ImageRegion< int32, 3 > input, M4D::Imaging::ImageRegion< int32, 3 > output, int32 threshold );
template void Sobel3D( M4D::Imaging::ImageRegion< uint32, 3 > input, M4D::Imaging::ImageRegion< uint32, 3 > output, uint32 threshold );
template void Sobel3D( M4D::Imaging::ImageRegion< int64, 3 > input, M4D::Imaging::ImageRegion< int64, 3 > output, int64 threshold );
template void Sobel3D( M4D::Imaging::ImageRegion< uint64, 3 > input, M4D::Imaging::ImageRegion< uint64, 3 > output, uint64 threshold );
template void Sobel3D( M4D::Imaging::ImageRegion< float, 3 > input, M4D::Imaging::ImageRegion< float, 3 > output, float threshold );
template void Sobel3D( M4D::Imaging::ImageRegion< double, 3 > input, M4D::Imaging::ImageRegion< double, 3 > output, double threshold );

template void LocalMinima3D( M4D::Imaging::ImageRegion< int8, 3 > input, M4D::Imaging::MaskRegion3D output );
template void LocalMinima3D( M4D::Imaging::ImageRegion< uint8, 3 > input, M4D::Imaging::MaskRegion3D output );
template void LocalMinima3D( M4D::Imaging::ImageRegion< int16, 3 > input, M4D::Imaging::MaskRegion3D output );
template void LocalMinima3D( M4D::Imaging::ImageRegion< uint16, 3 > input, M4D::Imaging::MaskRegion3D output );
template void LocalMinima3D( M4D::Imaging::ImageRegion< int32, 3 > input, M4D::Imaging::MaskRegion3D output );
template void LocalMinima3D( M4D::Imaging::ImageRegion< uint32, 3 > input, M4D::Imaging::MaskRegion3D output );
template void LocalMinima3D( M4D::Imaging::ImageRegion< int64, 3 > input, M4D::Imaging::MaskRegion3D output );
template void LocalMinima3D( M4D::Imaging::ImageRegion< uint64, 3 > input, M4D::Imaging::MaskRegion3D output );
template void LocalMinima3D( M4D::Imaging::ImageRegion< float, 3 > input, M4D::Imaging::MaskRegion3D output );
template void LocalMinima3D( M4D::Imaging::ImageRegion< double, 3 > input, M4D::Imaging::MaskRegion3D output );

template void WatershedTransformation3D( M4D::Imaging::ImageRegion< uint32, 3 > aLabeledMarkerRegions, M4D::Imaging::ImageRegion< int8, 3 > aInput, M4D::Imaging::ImageRegion< uint32, 3 > aOutput );
template void WatershedTransformation3D( M4D::Imaging::ImageRegion< uint32, 3 > aLabeledMarkerRegions, M4D::Imaging::ImageRegion< uint8, 3 > aInput, M4D::Imaging::ImageRegion< uint32, 3 > aOutput );
template void WatershedTransformation3D( M4D::Imaging::ImageRegion< uint32, 3 > aLabeledMarkerRegions, M4D::Imaging::ImageRegion< int16, 3 > aInput, M4D::Imaging::ImageRegion< uint32, 3 > aOutput );
template void WatershedTransformation3D( M4D::Imaging::ImageRegion< uint32, 3 > aLabeledMarkerRegions, M4D::Imaging::ImageRegion< uint16, 3 > aInput, M4D::Imaging::ImageRegion< uint32, 3 > aOutput );
template void WatershedTransformation3D( M4D::Imaging::ImageRegion< uint32, 3 > aLabeledMarkerRegions, M4D::Imaging::ImageRegion< int32, 3 > aInput, M4D::Imaging::ImageRegion< uint32, 3 > aOutput );
template void WatershedTransformation3D( M4D::Imaging::ImageRegion< uint32, 3 > aLabeledMarkerRegions, M4D::Imaging::ImageRegion< uint32, 3 > aInput, M4D::Imaging::ImageRegion< uint32, 3 > aOutput );
template void WatershedTransformation3D( M4D::Imaging::ImageRegion< uint32, 3 > aLabeledMarkerRegions, M4D::Imaging::ImageRegion< int64, 3 > aInput, M4D::Imaging::ImageRegion< uint32, 3 > aOutput );
template void WatershedTransformation3D( M4D::Imaging::ImageRegion< uint32, 3 > aLabeledMarkerRegions, M4D::Imaging::ImageRegion< uint64, 3 > aInput, M4D::Imaging::ImageRegion< uint32, 3 > aOutput );
template void WatershedTransformation3D( M4D::Imaging::ImageRegion< uint32, 3 > aLabeledMarkerRegions, M4D::Imaging::ImageRegion< float, 3 > aInput, M4D::Imaging::ImageRegion< uint32, 3 > aOutput );
template void WatershedTransformation3D( M4D::Imaging::ImageRegion< uint32, 3 > aLabeledMarkerRegions, M4D::Imaging::ImageRegion< double, 3 > aInput, M4D::Imaging::ImageRegion< uint32, 3 > aOutput );
