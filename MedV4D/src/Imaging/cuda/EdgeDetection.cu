#include "hip/hip_runtime.h"
#include "MedV4D/Imaging/cuda/detail/EdgeDetection.cuh"


template< typename RegionType >
void
Sobel3D( RegionType input, RegionType output, typename RegionType::ElementType threshold )
{
	typedef typename RegionType::ElementType TElement;
	typedef Buffer3D< TElement > Buffer;

	Buffer inBuffer = CudaBuffer3DFromImageRegionCopy( input );
	Buffer outBuffer = CudaBuffer3DFromImageRegion( output );

	SobelFilter3DFtor< TElement > filter( threshold );
	//int3 radius = filter.radius;

	dim3 blockSize( 8, 8, 8 );
	int3 blockResolution = GetBlockResolution( inBuffer.mSize, blockSize, make_int3(0,0,0) );
	dim3 gridSize( blockResolution.x * blockResolution.y, blockResolution.z, 1 );

	M4D::Common::Clock clock;
	CheckCudaErrorState( "Before kernel execution" );
	FilterKernel3D< TElement, TElement, SobelFilter3DFtor< TElement > >
		<<< gridSize, blockSize >>>( 
					inBuffer, 
					outBuffer, 
					blockResolution,
					filter
					);
	hipDeviceSynchronize();
	CheckCudaErrorState( "After kernel execution" );
	LOG( "Sobel3D computations took " << clock.SecondsPassed() )

	hipMemcpy(output.GetPointer(), outBuffer.mData, outBuffer.mLength * sizeof(TElement), hipMemcpyDeviceToHost );
	CheckCudaErrorState( "Copy back" );
	//hipFree( inBuffer.mData );
	//hipFree( outBuffer.mData );
	CheckCudaErrorState( "Free memory" );
}

#define DECLARE_TEMPLATE_INSTANCE template void Sobel3D( M4D::Imaging::ImageRegion< TTYPE, 3 > input, M4D::Imaging::ImageRegion< TTYPE, 3 > output, TTYPE threshold );
#include "MedV4D/Common/DeclareTemplateNumericInstances.h"

